#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xsumsa01
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(idx >= N)
        return;

    float vel_x = 0;
    float vel_y = 0;
    float vel_z = 0;

    for(int i = 0; i < N; i++) {
        float r, dx, dy, dz;

        dx = p_in.pos_x[i] - p_in.pos_x[idx];
        dy = p_in.pos_y[i] - p_in.pos_y[idx];
        dz = p_in.pos_z[i] - p_in.pos_z[idx];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        if(r > COLLISION_DISTANCE) {
            /* Newton's law of universal gravitation:
             *      F = G * ((m1 * m2) / r^2) * u
             * where G is the gravitational constant, m1 and m2 are masses of particles,
             * r is distance, and u is a unit vector defined as:
             *      u = (r2 - r1) / r
             *
             * Gravitational velocity:
             *      v_g = F / m * d_t
             */

            float f = (G * p_in.weight[idx] * p_in.weight[i]) / (r * r);

            vel_x += r != 0.0f ? (((f * (dx/r)) / p_in.weight[idx]) * dt) : 0.0f;
            vel_y += r != 0.0f ? (((f * (dy/r)) / p_in.weight[idx]) * dt) : 0.0f;
            vel_z += r != 0.0f ? (((f * (dz/r)) / p_in.weight[idx]) * dt) : 0.0f;
        } else if(r > 0.0f && r < COLLISION_DISTANCE) {
            /* Collision velocities:
             *      w1 = (m1 - m2) * v1 / M + 2 * m2 * v2 / M
             *  where m1 and m2 are masses of particles, v1 and v2 are velocities, and
             *  M is the center of mass calculated as m1 + m2
             */

            float mtot = p_in.weight[idx] + p_in.weight[i];
            float mdif = p_in.weight[idx] - p_in.weight[i];

            vel_x += ((mdif * p_in.vel_x[idx] / mtot) + 2 * (p_in.weight[i] * p_in.vel_x[i]) / mtot) - p_in.vel_x[idx];
            vel_y += ((mdif * p_in.vel_y[idx] / mtot) + 2 * (p_in.weight[i] * p_in.vel_y[i]) / mtot) - p_in.vel_y[idx];
            vel_z += ((mdif * p_in.vel_z[idx] / mtot) + 2 * (p_in.weight[i] * p_in.vel_z[i]) / mtot) - p_in.vel_z[idx];
        }
    }

    p_out.vel_x[idx] = p_in.vel_x[idx] + vel_x;
    p_out.vel_y[idx] = p_in.vel_y[idx] + vel_y;
    p_out.vel_z[idx] = p_in.vel_z[idx] + vel_z;

    p_out.pos_x[idx] = p_in.pos_x[idx] + p_out.vel_x[idx] * dt;
    p_out.pos_y[idx] = p_in.pos_y[idx] + p_out.vel_y[idx] * dt;
    p_out.pos_z[idx] = p_in.pos_z[idx] + p_out.vel_z[idx] * dt;
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
    for(int i = 0; i < N; i++) {
        fscanf(fp, "%f %f %f %f %f %f %f \n",
                &p.pos_x[i], &p.pos_y[i], &p.pos_z[i],
                &p.vel_x[i], &p.vel_y[i], &p.vel_z[i],
                &p.weight[i]);
    }
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
                p.pos_x[i], p.pos_y[i], p.pos_z[i],
                p.vel_x[i], p.vel_y[i], p.vel_z[i],
                p.weight[i]);
    }
}
