#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xsumsa01
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(idx >= N)
        return;

    for(int i = 0; i < N; i++) {
        float r, dx, dy, dz;
        float vx, vy, vz;

        dx = p.pos_x[i] - p.pos_x[idx];
        dy = p.pos_y[i] - p.pos_y[idx];
        dz = p.pos_z[i] - p.pos_z[idx];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        if(r > COLLISION_DISTANCE) {
            /* Newton's law of universal gravitation:
             *      F = G * ((m1 * m2) / r^2) * u
             * where G is the gravitational constant, m1 and m2 are masses of particles,
             * r is distance, and u is a unit vector defined as:
             *      u = (r2 - r1) / r
             *
             * Gravitational velocity:
             *      v_g = F / m * d_t
             */

            float f = (G * p.weight[idx] * p.weight[i]) / (r * r);

            vx = r != 0.0f ? (((f * (dx/r)) / p.weight[idx]) * dt) : 0.0f;
            vy = r != 0.0f ? (((f * (dy/r)) / p.weight[idx]) * dt) : 0.0f;
            vz = r != 0.0f ? (((f * (dz/r)) / p.weight[idx]) * dt) : 0.0f;

            tmp_vel.x[idx] += vx;
            tmp_vel.y[idx] += vy;
            tmp_vel.z[idx] += vz;
        }
    }
}

__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(idx >= N)
        return;

    for(int i = 0; i < N; i++) {
        float r, dx, dy, dz;
        float vx, vy, vz;

        dx = p.pos_x[i] - p.pos_x[idx];
        dy = p.pos_y[i] - p.pos_y[idx];
        dz = p.pos_z[i] - p.pos_z[idx];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        if(r > 0.0f && r < COLLISION_DISTANCE) {
            /* Collision velocities:
             *      w1 = (m1 - m2) * v1 / M + 2 * m2 * v2 / M
             *  where m1 and m2 are masses of particles, v1 and v2 are velocities, and
             *  M is the center of mass calculated as m1 + m2
             */

            float mtot = p.weight[idx] + p.weight[i];
            float mdif = p.weight[idx] - p.weight[i];

            vx = ((mdif * p.vel_x[idx] / mtot) + 2 * (p.weight[i] * p.vel_x[i]) / mtot) - p.vel_x[idx];
            vy = ((mdif * p.vel_y[idx] / mtot) + 2 * (p.weight[i] * p.vel_y[i]) / mtot) - p.vel_y[idx];
            vz = ((mdif * p.vel_z[idx] / mtot) + 2 * (p.weight[i] * p.vel_z[i]) / mtot) - p.vel_z[idx];

            tmp_vel.x[idx] += vx;
            tmp_vel.y[idx] += vy;
            tmp_vel.z[idx] += vz;
        }
    }
}

__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(idx >= N)
        return;

    p.vel_x[idx] += tmp_vel.x[idx];
    p.vel_y[idx] += tmp_vel.y[idx];
    p.vel_z[idx] += tmp_vel.z[idx];

    p.pos_x[idx] += p.vel_x[idx] * dt;
    p.pos_y[idx] += p.vel_y[idx] * dt;
    p.pos_z[idx] += p.vel_z[idx] * dt;
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
    for(int i = 0; i < N; i++) {
        fscanf(fp, "%f %f %f %f %f %f %f \n",
                &p.pos_x[i], &p.pos_y[i], &p.pos_z[i],
                &p.vel_x[i], &p.vel_y[i], &p.vel_z[i],
                &p.weight[i]);
    }
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
                p.pos_x[i], p.pos_y[i], p.pos_z[i],
                p.vel_x[i], p.vel_y[i], p.vel_z[i],
                p.weight[i]);
    }
}
