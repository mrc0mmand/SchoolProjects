/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xsumsa01
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"

#define NF(x) (N * sizeof(float))
/**
  * @brief Allocate memory on CPU
  *
  * @param t Data type of the allocated memory
  * @param x Destination pointer
  * @param s Size of the allocated memory
  */
#define CPU_ALLOC(t, x, s) \
    do { \
        if(hipHostMalloc(&x, sizeof(t) * (s)) != hipSuccess) { \
            hipError_t e = hipGetLastError(); \
            fprintf(stderr, "hipHostMalloc() failed: %s\n", hipGetErrorString(e)); \
            exit(EXIT_FAILURE); \
        } \
        hipMemset(x, 0, N * sizeof(*x)); \
    } while(0)

#define CPU_FREE(x) \
    do { \
        hipHostFree(x); \
        x = NULL; \
    } while(0)

#define GPU_ALLOC(t, x, s) \
    do { \
        if(hipMalloc(&x, sizeof(t) * (s)) != hipSuccess) { \
            hipError_t e = hipGetLastError(); \
            fprintf(stderr, "hipMalloc() failed: %s\n", hipGetErrorString(e)); \
            exit(EXIT_FAILURE); \
        } \
        hipMemset(x, 0, N * sizeof(*x)); \
    } while(0)

#define GPU_FREE(x) \
    do { \
        hipFree(x); \
        x = NULL; \
    } while(0)

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;

    CPU_ALLOC(float, particles_cpu.pos_x, N);
    CPU_ALLOC(float, particles_cpu.pos_y, N);
    CPU_ALLOC(float, particles_cpu.pos_z, N);
    CPU_ALLOC(float, particles_cpu.vel_x, N);
    CPU_ALLOC(float, particles_cpu.vel_y, N);
    CPU_ALLOC(float, particles_cpu.vel_z, N);
    CPU_ALLOC(float, particles_cpu.weight, N);

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu;
    t_velocities tmp_velocities_gpu;

    /* DOPLNTE: alokaci pameti na GPU */
    GPU_ALLOC(float, particles_gpu.pos_x, N);
    GPU_ALLOC(float, particles_gpu.pos_y, N);
    GPU_ALLOC(float, particles_gpu.pos_z, N);
    GPU_ALLOC(float, particles_gpu.vel_x, N);
    GPU_ALLOC(float, particles_gpu.vel_y, N);
    GPU_ALLOC(float, particles_gpu.vel_z, N);
    GPU_ALLOC(float, particles_gpu.weight, N);

    GPU_ALLOC(float, tmp_velocities_gpu.x, N);
    GPU_ALLOC(float, tmp_velocities_gpu.y, N);
    GPU_ALLOC(float, tmp_velocities_gpu.z, N);

    hipMemcpy(particles_gpu.pos_x, particles_cpu.pos_x, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.pos_y, particles_cpu.pos_y, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.pos_z, particles_cpu.pos_z, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel_x, particles_cpu.vel_x, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel_y, particles_cpu.vel_y, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel_z, particles_cpu.vel_z, NF(N), hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.weight, particles_cpu.weight, NF(N), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "hipMemcpy(): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    size_t grid = (N / thr_blc) + 1; //(N + thr_blc- 1) / thr_blc;

    // vypocet
    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; ++s)
    {
        calculate_gravitation_velocity<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
        calculate_collision_velocity<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
        update_particle<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
        hipMemset(tmp_velocities_gpu.x, 0, NF(N));
        hipMemset(tmp_velocities_gpu.y, 0, NF(N));
        hipMemset(tmp_velocities_gpu.z, 0, NF(N));
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    hipMemcpy(particles_cpu.pos_x, particles_gpu.pos_x, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.pos_y, particles_gpu.pos_y, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.pos_z, particles_gpu.pos_z, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_x, particles_gpu.vel_x, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_y, particles_gpu.vel_y, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_z, particles_gpu.vel_z, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.weight, particles_gpu.weight, NF(N), hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

    // Cleanup
    CPU_FREE(particles_cpu.pos_x);
    CPU_FREE(particles_cpu.pos_y);
    CPU_FREE(particles_cpu.pos_z);
    CPU_FREE(particles_cpu.vel_x);
    CPU_FREE(particles_cpu.vel_y);
    CPU_FREE(particles_cpu.vel_z);
    CPU_FREE(particles_cpu.weight);

    GPU_FREE(particles_gpu.pos_x);
    GPU_FREE(particles_gpu.pos_y);
    GPU_FREE(particles_gpu.pos_z);
    GPU_FREE(particles_gpu.vel_x);
    GPU_FREE(particles_gpu.vel_y);
    GPU_FREE(particles_gpu.vel_z);
    GPU_FREE(particles_gpu.weight);

    GPU_FREE(tmp_velocities_gpu.x);
    GPU_FREE(tmp_velocities_gpu.y);
    GPU_FREE(tmp_velocities_gpu.z);

    return 0;
}
