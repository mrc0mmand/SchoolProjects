/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xsumsa01
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"

#define NF(x) (N * sizeof(float))

/**
  * @brief Allocate memory on CPU
  *
  * @param t Data type of the allocated memory
  * @param x Destination pointer
  * @param s Size of the allocated memory
  */
#define CPU_ALLOC(t, x, s) \
    do { \
        if(hipHostMalloc(&x, sizeof(t) * (s)) != hipSuccess) { \
            hipError_t e = hipGetLastError(); \
            fprintf(stderr, "hipHostMalloc() failed: %s\n", hipGetErrorString(e)); \
            exit(EXIT_FAILURE); \
        } \
        hipMemset(x, 0, N * sizeof(*x)); \
    } while(0)

/**
  * @brief Free memory allocated by CPU_ALLOC
  *
  * @param x Pointer to allocated memory
  */
#define CPU_FREE(x) \
    do { \
        hipHostFree(x); \
        x = NULL; \
    } while(0)

/**
  * @brief Allocate memory on GPU
  *
  * @param t Data type of the allocated memory
  * @param x Destination pointer
  * @param s Size of the allocated memory
  */
#define GPU_ALLOC(t, x, s) \
    do { \
        if(hipMalloc(&x, sizeof(t) * (s)) != hipSuccess) { \
            hipError_t e = hipGetLastError(); \
            fprintf(stderr, "hipMalloc() failed: %s\n", hipGetErrorString(e)); \
            exit(EXIT_FAILURE); \
        } \
        hipMemset(x, 0, N * sizeof(*x)); \
    } while(0)

/**
  * @brief Free memory allocated by GPU_ALLOC
  *
  * @param x Pointer to allocated memory
  */
#define GPU_FREE(x) \
    do { \
        hipFree(x); \
        x = NULL; \
    } while(0)

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;

    CPU_ALLOC(float, particles_cpu.pos_x, N);
    CPU_ALLOC(float, particles_cpu.pos_y, N);
    CPU_ALLOC(float, particles_cpu.pos_z, N);
    CPU_ALLOC(float, particles_cpu.vel_x, N);
    CPU_ALLOC(float, particles_cpu.vel_y, N);
    CPU_ALLOC(float, particles_cpu.vel_z, N);
    CPU_ALLOC(float, particles_cpu.weight, N);

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu[2];

    /* DOPLNTE: alokaci pameti na GPU */
    for(size_t i = 0; i < 2; i++) {
        GPU_ALLOC(float, particles_gpu[i].pos_x, N);
        GPU_ALLOC(float, particles_gpu[i].pos_y, N);
        GPU_ALLOC(float, particles_gpu[i].pos_z, N);
        GPU_ALLOC(float, particles_gpu[i].vel_x, N);
        GPU_ALLOC(float, particles_gpu[i].vel_y, N);
        GPU_ALLOC(float, particles_gpu[i].vel_z, N);
        GPU_ALLOC(float, particles_gpu[i].weight, N);

        hipMemcpy(particles_gpu[i].pos_x, particles_cpu.pos_x, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].pos_y, particles_cpu.pos_y, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].pos_z, particles_cpu.pos_z, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].vel_x, particles_cpu.vel_x, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].vel_y, particles_cpu.vel_y, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].vel_z, particles_cpu.vel_z, NF(N), hipMemcpyHostToDevice);
        hipMemcpy(particles_gpu[i].weight, particles_cpu.weight, NF(N), hipMemcpyHostToDevice);
    }

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "hipMemcpy(): %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    size_t grid = (N / thr_blc) + 1; //(N + thr_blc- 1) / thr_blc;

    // vypocet
    gettimeofday(&t1, 0);

    size_t p_in_idx = 0;
    size_t p_out_idx = 0;
    size_t sh_mem = thr_blc * sizeof(float) * P_EN_SIZE;
    for (int s = 0; s < steps; ++s)
    {
        // Swap p_in and p_out in each step
        p_in_idx = s % 2;
        p_out_idx = (s + 1) % 2;
        calculate_velocity<<<grid, thr_blc, sh_mem>>>(particles_gpu[p_in_idx], particles_gpu[p_out_idx], N, dt);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    hipMemcpy(particles_cpu.pos_x, particles_gpu[p_out_idx].pos_x, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.pos_y, particles_gpu[p_out_idx].pos_y, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.pos_z, particles_gpu[p_out_idx].pos_z, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_x, particles_gpu[p_out_idx].vel_x, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_y, particles_gpu[p_out_idx].vel_y, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel_z, particles_gpu[p_out_idx].vel_z, NF(N), hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.weight, particles_gpu[p_out_idx].weight, NF(N), hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

    // Cleanup
    CPU_FREE(particles_cpu.pos_x);
    CPU_FREE(particles_cpu.pos_y);
    CPU_FREE(particles_cpu.pos_z);
    CPU_FREE(particles_cpu.vel_x);
    CPU_FREE(particles_cpu.vel_y);
    CPU_FREE(particles_cpu.vel_z);
    CPU_FREE(particles_cpu.weight);

    for(size_t i = 0; i < 2; i++) {
        GPU_FREE(particles_gpu[0].pos_x);
        GPU_FREE(particles_gpu[0].pos_y);
        GPU_FREE(particles_gpu[0].pos_z);
        GPU_FREE(particles_gpu[0].vel_x);
        GPU_FREE(particles_gpu[0].vel_y);
        GPU_FREE(particles_gpu[0].vel_z);
        GPU_FREE(particles_gpu[0].weight);
    }

    return 0;
}
