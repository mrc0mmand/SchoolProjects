#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xsumsa01
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
  * @brief Calculate index to the shared memory
  * @details Calculation: thread_id * #t_particles + offset,
             where thread_idx is ID of the active thread,
             #t_particles is number of items of the t_particles structure,
             and offset is index of each structure member defined by
             enum in nbody.h file
  *
  * @param id Thread ID
  * @param x Offset (see enum in nbody.h)
  */
#define SH_IDX(id, x) (id * P_EN_SIZE + (x))

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int tid = threadIdx.x;
    extern __shared__ float p_sh[];

    if(idx >= N)
        return;

    p_sh[SH_IDX(tid, POS_X)] = p_in.pos_x[idx];
    p_sh[SH_IDX(tid, POS_Y)] = p_in.pos_y[idx];
    p_sh[SH_IDX(tid, POS_Z)] = p_in.pos_z[idx];
    p_sh[SH_IDX(tid, VEL_X)] = p_in.vel_x[idx];
    p_sh[SH_IDX(tid, VEL_Y)] = p_in.vel_y[idx];
    p_sh[SH_IDX(tid, VEL_Z)] = p_in.vel_z[idx];
    p_sh[SH_IDX(tid, WEIGHT)] = p_in.weight[idx];
    __syncthreads();

    float vel_x = 0;
    float vel_y = 0;
    float vel_z = 0;

    for(int i = 0; i < N; i++) {
        float r, dx, dy, dz;

        dx = p_in.pos_x[i] - p_sh[SH_IDX(tid, POS_X)];
        dy = p_in.pos_y[i] - p_sh[SH_IDX(tid, POS_Y)];
        dz = p_in.pos_z[i] - p_sh[SH_IDX(tid, POS_Z)];

        r = sqrt(dx*dx + dy*dy + dz*dz);

        if(r > COLLISION_DISTANCE) {
            /* Newton's law of universal gravitation:
             *      F = G * ((m1 * m2) / r^2) * u
             * where G is the gravitational constant, m1 and m2 are masses of particles,
             * r is distance, and u is a unit vector defined as:
             *      u = (r2 - r1) / r
             *
             * Gravitational velocity:
             *      v_g = F / m * d_t
             */

            float f = (G * p_sh[SH_IDX(tid, WEIGHT)] * p_in.weight[i]) / (r * r);

            vel_x += r != 0.0f ? (((f * (dx/r)) / p_sh[SH_IDX(tid, WEIGHT)]) * dt) : 0.0f;
            vel_y += r != 0.0f ? (((f * (dy/r)) / p_sh[SH_IDX(tid, WEIGHT)]) * dt) : 0.0f;
            vel_z += r != 0.0f ? (((f * (dz/r)) / p_sh[SH_IDX(tid, WEIGHT)]) * dt) : 0.0f;
        } else if(r > 0.0f && r < COLLISION_DISTANCE) {
            /* Collision velocities:
             *      w1 = (m1 - m2) * v1 / M + 2 * m2 * v2 / M
             *  where m1 and m2 are masses of particles, v1 and v2 are velocities, and
             *  M is the center of mass calculated as m1 + m2
             */

            float mtot = p_sh[SH_IDX(tid, WEIGHT)] + p_in.weight[i];
            float mdif = p_sh[SH_IDX(tid, WEIGHT)] - p_in.weight[i];

            vel_x += ((mdif * p_sh[SH_IDX(tid, VEL_X)] / mtot) + 2 * (p_in.weight[i] * p_in.vel_x[i]) / mtot) - p_sh[SH_IDX(tid, VEL_X)];
            vel_y += ((mdif * p_sh[SH_IDX(tid, VEL_Y)] / mtot) + 2 * (p_in.weight[i] * p_in.vel_y[i]) / mtot) - p_sh[SH_IDX(tid, VEL_Y)];
            vel_z += ((mdif * p_sh[SH_IDX(tid, VEL_Z)] / mtot) + 2 * (p_in.weight[i] * p_in.vel_z[i]) / mtot) - p_sh[SH_IDX(tid, VEL_Z)];
        }
    }

    p_out.vel_x[idx] = p_sh[SH_IDX(tid, VEL_X)] + vel_x;
    p_out.vel_y[idx] = p_sh[SH_IDX(tid, VEL_Y)] + vel_y;
    p_out.vel_z[idx] = p_sh[SH_IDX(tid, VEL_Z)] + vel_z;

    p_out.pos_x[idx] = p_sh[SH_IDX(tid, POS_X)] + p_out.vel_x[idx] * dt;
    p_out.pos_y[idx] = p_sh[SH_IDX(tid, POS_Y)] + p_out.vel_y[idx] * dt;
    p_out.pos_z[idx] = p_sh[SH_IDX(tid, POS_Z)] + p_out.vel_z[idx] * dt;
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
    for(int i = 0; i < N; i++) {
        fscanf(fp, "%f %f %f %f %f %f %f \n",
                &p.pos_x[i], &p.pos_y[i], &p.pos_z[i],
                &p.vel_x[i], &p.vel_y[i], &p.vel_z[i],
                &p.weight[i]);
    }
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
    for (int i = 0; i < N; i++)
    {
        fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
                p.pos_x[i], p.pos_y[i], p.pos_z[i],
                p.vel_x[i], p.vel_y[i], p.vel_z[i],
                p.weight[i]);
    }
}
